/* ****************************************************************************************************************** */
/**
 *  @file matrix_gpu.cu
 *  @author Edward J. Parkinson (e.parkinson@soton.ac.uk)
 *  @date August 2023
 *
 *  @brief
 *
 *  ***************************************************************************************************************** */


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

#if CUDA_ON

#include <cuda.h>
#include <cuda_runtime.h>
#include <cusolverDn.h>

/* NVCC is a C++ compiler at heart, so anything we re-use from regular C source
   has to be defined here with `extern "C"` to tell the compiler that the
   function has been compiled by a C compiler (and does some computer science
   stuff to make linking possible) */

extern "C" int Exit (int error_code);
extern "C" int Error (const char *format, ...);
extern "C" int Log (const char *format, ...);

/* `cusolver_handle` is a variable used to interact with the cuSolver/CUDA
    runtime and is used to initialise and clean up the resources required for
    both runtimes */

static cusolverDnHandle_t cusolver_handle = NULL;

/* ****************************************************************************************************************** */
/**
 *  @brief Check the return status of a CUDA function
 *
 *  @param [in] status  the status to check
 *
 *  ***************************************************************************************************************** */

#define CUDA_CHECK(status)                                                                                             \
  do {                                                                                                                 \
    cudaError_t err = status;                                                                                          \
    if (err != cudaSuccess) {                                                                                          \
      Error("CUDA Error: %s\n", cudaGetErrorString(err));                                                              \
      Exit(EXIT_FAILURE);                                                                                              \
    }                                                                                                                  \
  } while (0)

/* ****************************************************************************************************************** */
/**
 *  @brief Check the return status of a cuSOLVER function
 *
 *  @param [in] status  the status to check
 *
 *  ***************************************************************************************************************** */

#define CUSOLVER_CHECK(status)                                                                                         \
  do {                                                                                                                 \
    cusolverStatus_t err = status;                                                                                     \
    if (err != CUSOLVER_STATUS_SUCCESS) {                                                                              \
      Error("cuSolver Error: %d\n", err);                                                                              \
      Exit(EXIT_FAILURE);                                                                                              \
    }                                                                                                                  \
  } while (0)

/* ****************************************************************************************************************** */
/**
 * @brief
 *
 * @return
 *
 * @details
 *
 *  ***************************************************************************************************************** */

extern "C" void
cuda_init (void)
{
  CUSOLVER_CHECK (cusolverDnCreate (&cusolver_handle));
  Log ("Created a new cuSOLVER handle created\n");
}

/* ****************************************************************************************************************** */
/**
 * @brief
 *
 * @return
 *
 * @details
 *
 *  ***************************************************************************************************************** */

extern "C" void
cuda_finish (void)
{
  CUSOLVER_CHECK (cusolverDnDestroy (cusolver_handle));
  Log ("Destroyed the cuSOLVER handle\n");
}

/* ****************************************************************************************************************** */
/**
 * @brief
 *
 * @return
 *
 * @details
 *
 *  ***************************************************************************************************************** */

__global__ void
createIdentityMatrixKernel (double *d_identity, int size)
{
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;

  if (row < size && col < size)
  {
    d_identity[row * size + col] = (row == col) ? 1.0 : 0.0;
  }
}

/* ****************************************************************************************************************** */
/**
 * @brief  Solve the linear system A x = b, for the vector x
 *
 * @param  [in]  a_matrix  a square matrix on the LHS
 * @param  [in]  b_vector  the B resultant vector
 * @param  [in]  size  the number of rows (and columns) in the square matrix matrix and vectors
 * @param  [out] x_vector  the x vector on the RHS
 *
 * @return an integer representing the error state
 *
 * @details
 * Performs LU decomposition to solve for x in the linear system A x = b. The calculation is perform, in serial, on
 * the CPU using GSL.
 *
 *  ***************************************************************************************************************** */

extern "C" int
gpu_solve_linear_system (double *a_matrix, double *b_vector, int size, double *x_vector)
{
  int *devInfo;
  int lwork;
  int *d_pivot;                 /* device array of pivoting sequence */
  double *d_A, *d_b;
  double *d_work;               /* cuSolver needs a "workspace" to do stuff, which we have to allocate manually */

  /* Allocate memory on the GPU (device) to store the matrices/vectors */
  cudaMalloc ((void **) &d_A, size * size * sizeof (double));
  cudaMalloc ((void **) &d_b, size * sizeof (double));
  cudaMalloc ((void **) &devInfo, sizeof (int));

  /* Copy the matrix and vector to the device memory */
  cudaMemcpy (d_A, a_matrix, size * size * sizeof (double), cudaMemcpyHostToDevice);
  cudaMemcpy (d_b, b_vector, size * sizeof (double), cudaMemcpyHostToDevice);
  cudaMalloc ((void **) &d_pivot, size * sizeof (int));

  /* XXXX_bufferSize is used to compute the size of the workspace we need, and depends on the size of the linear
     system being solved */
  cusolverDnDgetrf_bufferSize (cusolver_handle, size, size, d_A, size, &lwork);
  cudaMalloc ((void **) &d_work, lwork * sizeof (double));

  /* Perform LU factorization and solve the linear system. The vector d_b is not used in `getrs` (the solver), but
     it's the same size of the solution vector so we'll re-use that. d_b is then copied back to host memory (CPU RAM) */
  cusolverDnDgetrf (cusolver_handle, size, size, d_A, size, d_work, d_pivot, devInfo);
  cusolverDnDgetrs (cusolver_handle, CUBLAS_OP_T, size, 1, d_A, size, d_pivot, d_b, size, devInfo);
  cudaMemcpy (x_vector, d_b, size * sizeof (double), cudaMemcpyDeviceToHost);

  cudaFree (d_A);
  cudaFree (d_b);
  cudaFree (d_work);
  cudaFree (d_pivot);

  return EXIT_SUCCESS;
}

/* ****************************************************************************************************************** */
/**
 * @brief
 *
 * @param  [in]  a_matrix
 * @param  [out] a_inverse
 * @param  [in]  num_rows
 *
 * @return an integer representing the error state
 *
 * @details
 *
 *  ***************************************************************************************************************** */

extern "C" int
gpu_invert_matrix (double *matrix, double *inverse_matrix, int num_rows)
{
  int *d_pivot;
  int work_size;
  int *dev_info;
  double *d_matrix;
  double *d_identity;
  double *d_workspace;

  cudaMalloc ((void **) &d_matrix, num_rows * num_rows * sizeof (double));
  cudaMalloc ((void **) &d_pivot, num_rows * sizeof (int));
  cudaMalloc ((void **) &d_identity, num_rows * num_rows * sizeof (double));
  cudaMalloc ((void **) &dev_info, sizeof (int));

  cudaMemcpy (d_matrix, matrix, num_rows * num_rows * sizeof (double), cudaMemcpyHostToDevice);

  dim3 blockSize (16, 16);
  dim3 gridSize ((num_rows + blockSize.x - 1) / blockSize.x, (num_rows + blockSize.y - 1) / blockSize.y);
  createIdentityMatrixKernel <<< gridSize, blockSize >>> (d_identity, num_rows);

  cusolverDnDgetrf_bufferSize (cusolver_handle, num_rows, num_rows, d_matrix, num_rows, &work_size);
  cudaMalloc ((void **) &d_workspace, work_size * sizeof (double));

  cusolverDnDgetrf (cusolver_handle, num_rows, num_rows, d_matrix, num_rows, d_workspace, d_pivot, dev_info);
  cusolverDnDgetrs (cusolver_handle, CUBLAS_OP_T, num_rows, num_rows, d_matrix, num_rows, d_pivot, d_identity, num_rows, dev_info);

  cudaMemcpy (inverse_matrix, d_identity, num_rows * num_rows * sizeof (double), cudaMemcpyDeviceToHost);

  cudaFree (d_matrix);
  cudaFree (d_identity);
  cudaFree (d_workspace);
  cudaFree (dev_info);

  return EXIT_SUCCESS;
}

#endif
