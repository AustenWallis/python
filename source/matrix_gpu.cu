#include "hip/hip_runtime.h"
/* ****************************************************************************************************************** */
/**
 *  @file matrix_gpu.cu
 *  @author Edward J. Parkinson (e.parkinson@soton.ac.uk)
 *  @date August 2023
 *
 *  @brief
 *
 *  ***************************************************************************************************************** */

#include <math.h>
#include <stdlib.h>
#include <stdio.h>

#include "log.h"

#if CUDA_ON

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

// #include "matrix.h"

hipsolverHandle_t cusolver_handle = NULL;

/* ****************************************************************************************************************** */
/**
 *  @brief Check the return status of a CUDA function
 *
 *  @param [in] status  the status to check
 *
 *  ***************************************************************************************************************** */

#define CUDA_CHECK(status)                                                                                             \
  do {                                                                                                                 \
    hipError_t err = status;                                                                                          \
    if (err != hipSuccess) {                                                                                          \
      printf((char *)"CUDA Error: %s\n", hipGetErrorString(err));                                                              \
      return(EXIT_FAILURE);                                                                                              \
    }                                                                                                                  \
  } while (0)

/* ****************************************************************************************************************** */
/**
 *  @brief Check the return status of a cuSOLVER function
 *
 *  @param [in] status  the status to check
 *
 *  ***************************************************************************************************************** */

#define CUSOLVER_CHECK(status)                                                                                         \
  do {                                                                                                                 \
    hipsolverStatus_t err = status;                                                                                     \
    if (err != HIPSOLVER_STATUS_SUCCESS) {                                                                              \
      printf((char *)"cuSolver Error: %d\n", err);                                                                              \
      return(EXIT_FAILURE);                                                                                              \
    }                                                                                                                  \
  } while (0)

/* ****************************************************************************************************************** */
/**
 * @brief  Solve the linear system A x = b, for the vector x
 *
 * @param  [in]  a_matrix  a square matrix on the LHS
 * @param  [in]  b_vector  the B resultant vector
 * @param  [in]  size  the number of rows (and columns) in the square matrix matrix and vectors
 * @param  [out] x_vector  the x vector on the RHS
 *
 * @return an integer representing the error state
 *
 * @details
 * Performs LU decomposition to solve for x in the linear system A x = b. The calculation is perform, in serial, on
 * the CPU using GSL.
 *
 *  ***************************************************************************************************************** */

extern "C" int
gpu_solve_linear_system (double *a_matrix, double *b_vector, int size, double *x_vector)
{
  if (cusolver_handle == NULL)
  {
    CUSOLVER_CHECK (hipsolverDnCreate (&cusolver_handle));
  }

  printf ((char *) "We are in the GPU function\n");

  // Device variables
  double *d_A, *d_b;
  int *devInfo;
  int lwork;
  double *d_work;

  // Allocate memory on the GPU
  hipMalloc ((void **) &d_A, size * size * sizeof (double));
  hipMalloc ((void **) &d_b, size * sizeof (double));
  hipMalloc ((void **) &devInfo, sizeof (int));

  // Transfer data to the GPU
  hipMemcpy (d_A, a_matrix, size * size * sizeof (double), hipMemcpyHostToDevice);
  hipMemcpy (d_b, b_vector, size * sizeof (double), hipMemcpyHostToDevice);

  // Perform LU factorization
  hipsolverDnDgetrf_bufferSize (cusolver_handle, size, size, d_A, size, &lwork);
  hipMalloc ((void **) &d_work, lwork * sizeof (double));

  int *d_pivot;                 // device array of pivoting sequence
  hipMalloc ((void **) &d_pivot, size * sizeof (int));

  hipsolverDnDgetrf (cusolver_handle, size, size, d_A, size, d_work, d_pivot, devInfo);

  // Solve the linear system
  // hipsolverDnDgetrs (cusolver_handle, HIPBLAS_OP_N, size, 1, d_A, size, d_pivot, d_b, size, devInfo);
  hipsolverDnDgetrs (cusolver_handle, HIPBLAS_OP_T, size, 1, d_A, size, d_pivot, d_b, size, devInfo);

  // Transfer the solution back to the host
  hipMemcpy (x_vector, d_b, size * sizeof (double), hipMemcpyDeviceToHost);

  // Clean up
  hipFree (d_A);
  hipFree (d_b);
  hipFree (d_work);
  hipFree (d_pivot);
  hipsolverDnDestroy (cusolver_handle);

  return EXIT_SUCCESS;
}

/* ****************************************************************************************************************** */
/**
 * @brief
 *
 * @param  [in]  a_matrix
 * @param  [out] a_inverse
 * @param  [in]  num_rows
 *
 * @return an integer representing the error state
 *
 * @details
 *
 *  ***************************************************************************************************************** */

extern "C" int
gpu_invert_matrix (double *matrix, double *inverse, int num_rows)
{
  if (cusolver_handle == NULL)
  {
    CUSOLVER_CHECK (hipsolverDnCreate (&cusolver_handle));
  }

  return EXIT_SUCCESS;
}

#endif
